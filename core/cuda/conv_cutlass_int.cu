#include <conv2d_NHWC.cuh>
#include <transform.cuh>
#include <cstdint>
#include <cstdlib>
#include <string>
// constexpr const char* PIGEON_LAYOUT = "NCHW"; // USE_CUDA_GEMM 2
// constexpr const char* PIGEON_LAYOUT = "CHWN"; // USE_CUDA_GEMM 4

template <typename Type>
void conv2d_cutlass(const Type* X,
                    const Type* W,
                    Type* Y,
                    int batchSize,
                    int inh,
                    int inw,
                    int din,
                    int dout,
                    int wh,
                    int ww,
                    int padding,
                    int stride,
                    int dilation = 1)
{
    Type* x;
    Type* w;
    Type* y;
    Type* xt;
    Type* wt;
    Type* yt;
    int xSize = inh * inw * din * batchSize;
    int wSize = wh * ww * din * dout;
    int outh = (inh + 2 * padding - wh - (wh - 1) * (dilation - 1)) / stride + 1;
    int outw = (inw + 2 * padding - ww - (ww - 1) * (dilation - 1)) / stride + 1;
    int ySize = outh * outw * dout * batchSize;
    hipMalloc((void**)&x, xSize * sizeof(Type));
    hipMemcpy(x, X, xSize * sizeof(Type), hipMemcpyHostToDevice);
    hipMalloc((void**)&xt, xSize * sizeof(Type));
    if constexpr (PIGEON_LAYOUT == "CHWN")
    {
        chwn_to_nhwc_(xt, x, batchSize, inh, inw, din);
    }
    else if constexpr (PIGEON_LAYOUT == "NCHW")
    {
        nchw_to_nhwc_(xt, x, batchSize, inh, inw, din);
    }
    hipFree(x);

    hipMalloc((void**)&w, wSize * sizeof(Type));
    hipMemcpy(w, W, wSize * sizeof(Type), hipMemcpyHostToDevice);
    hipMalloc((void**)&wt, wSize * sizeof(Type));
    if constexpr (PIGEON_LAYOUT != "NHWC")
    {
        nchw_to_nhwc_(wt, w, dout, wh, ww, din);
    }
    hipFree(w);

    hipMalloc((void**)&y, ySize * sizeof(Type));

    gpu::conv_fprop<Type>(xt, wt, y, batchSize, inh, inw, din, dout, wh, ww, padding, padding, stride, dilation);

    hipFree(xt);
    hipFree(wt);

    hipMalloc((void**)&yt, ySize * sizeof(Type));
    if constexpr (PIGEON_LAYOUT == "CHWN")
    {
        nhwc_to_chwn_(yt, y, batchSize, outh, outw, dout);
    }
    else if constexpr (PIGEON_LAYOUT == "NCHW")
    {
        nhwc_to_nchw_(yt, y, batchSize, outh, outw, dout);
    }

    hipMemcpy(Y, yt, ySize * sizeof(Type), hipMemcpyDeviceToHost);

    hipFree(yt);
}

// UINT8 and UINT16 are not supported by all architectures
// template void conv2d_cutlass<uint8_t>(const uint8_t* X, const uint8_t* W, uint8_t* Y, int batchSize, int inh, int
// inw, int din, int dout, int wh, int ww, int padding, int stride, int dilation);
template void conv2d_cutlass<uint16_t>(const uint16_t* X,
                                       const uint16_t* W,
                                       uint16_t* Y,
                                       int batchSize,
                                       int inh,
                                       int inw,
                                       int din,
                                       int dout,
                                       int wh,
                                       int ww,
                                       int padding,
                                       int stride,
                                       int dilation);  // INT8 and INT16 are not supported by all architectures
template void conv2d_cutlass<uint32_t>(const uint32_t* X,
                                       const uint32_t* W,
                                       uint32_t* Y,
                                       int batchSize,
                                       int inh,
                                       int inw,
                                       int din,
                                       int dout,
                                       int wh,
                                       int ww,
                                       int padding,
                                       int stride,
                                       int dilation);
template void conv2d_cutlass<uint64_t>(const uint64_t* X,
                                       const uint64_t* W,
                                       uint64_t* Y,
                                       int batchSize,
                                       int inh,
                                       int inw,
                                       int din,
                                       int dout,
                                       int wh,
                                       int ww,
                                       int padding,
                                       int stride,
                                       int dilation);
